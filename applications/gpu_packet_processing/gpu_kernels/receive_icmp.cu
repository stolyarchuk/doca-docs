#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023 NVIDIA CORPORATION AND AFFILIATES.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdlib.h>
#include <string.h>

#include <doca_gpunetio_dev_buf.cuh>
#include <doca_gpunetio_dev_eth_txq.cuh>
#include <doca_gpunetio_dev_eth_rxq.cuh>

#include "common.h"
#include "packets.h"
#include "filters.cuh"

DOCA_LOG_REGISTER(GPU_SANITY::KernelReceiveIcmp);

static
__device__ void icmp_swap_mac_addr(struct eth_ip_icmp_hdr *hdr)
{
	uint16_t addr_bytes[3];

	addr_bytes[0] = ((uint16_t *)hdr->l2_hdr.s_addr_bytes)[0];
	addr_bytes[1] = ((uint16_t *)hdr->l2_hdr.s_addr_bytes)[1];
	addr_bytes[2] = ((uint16_t *)hdr->l2_hdr.s_addr_bytes)[2];

	((uint16_t *)hdr->l2_hdr.s_addr_bytes)[0] = ((uint16_t *)hdr->l2_hdr.d_addr_bytes)[0];
	((uint16_t *)hdr->l2_hdr.s_addr_bytes)[1] = ((uint16_t *)hdr->l2_hdr.d_addr_bytes)[1];
	((uint16_t *)hdr->l2_hdr.s_addr_bytes)[2] = ((uint16_t *)hdr->l2_hdr.d_addr_bytes)[2];

	((uint16_t *)hdr->l2_hdr.d_addr_bytes)[0] = addr_bytes[0];
	((uint16_t *)hdr->l2_hdr.d_addr_bytes)[1] = addr_bytes[1];
	((uint16_t *)hdr->l2_hdr.d_addr_bytes)[2] = addr_bytes[2];
}

static
__device__ void icmp_swap_ip_addr(struct eth_ip_icmp_hdr *hdr)
{
	uint32_t tmp;

	tmp = hdr->l3_hdr.src_addr;
	hdr->l3_hdr.src_addr = hdr->l3_hdr.dst_addr;
	hdr->l3_hdr.dst_addr = tmp;
}

static __device__ uint16_t
icmp_checksum(const uint16_t *icmph, int len)
{
	uint32_t sum = 0;
	uint16_t odd_byte;

	while (len > 1) {
		sum += *icmph++;
		len -= 2;
	}

	if (len == 1) {
		*(uint8_t*)(&odd_byte) = * (uint8_t*)icmph;
		sum += odd_byte;
	}

	sum =  (sum >> 16) + (sum & 0xffff);
	sum += (sum >> 16);

	return (~sum);
}

__global__ void cuda_kernel_receive_icmp(uint32_t *exit_cond, struct doca_gpu_eth_rxq *rxq, struct doca_gpu_eth_txq *txq)
{
	__shared__ uint32_t rx_pkt_num;
	__shared__ uint64_t rx_buf_idx;

	doca_error_t ret;
	uint64_t buf_idx = 0;
	uintptr_t buf_addr;
	struct doca_gpu_buf *buf_ptr;
	struct eth_ip_icmp_hdr *hdr;
	uint8_t *payload;
	uint32_t nbytes;
	uint32_t lane_id = threadIdx.x % WARP_SIZE;
	uint32_t warp_id = threadIdx.x / WARP_SIZE;

	if (warp_id > 0)
		return;

	while (DOCA_GPUNETIO_VOLATILE(*exit_cond) == 0) {
		ret = doca_gpu_dev_eth_rxq_receive_warp(rxq, MAX_RX_NUM_PKTS_ICMP, MAX_RX_TIMEOUT_NS_ICMP, &rx_pkt_num, &rx_buf_idx);
		/* If any thread returns receive error, the whole execution stops */
		if (ret != DOCA_SUCCESS) {
			if (lane_id == 0) {
				/*
				 * printf in CUDA kernel may be a good idea only to report critical errors or debugging.
				 * If application prints this message on the console, something bad happened and
				 * applications needs to exit
				 */
				printf("Receive ICMP kernel error %d warp %d lane %d error %d\n", ret, warp_id, rx_pkt_num, ret);
				DOCA_GPUNETIO_VOLATILE(*exit_cond) = 1;
			}
			break;
		}

		if (rx_pkt_num == 0)
			continue;

		buf_idx = lane_id;
		while (buf_idx < rx_pkt_num) {
			doca_gpu_dev_eth_rxq_get_buf(rxq, rx_buf_idx + buf_idx, &buf_ptr);
			doca_gpu_dev_buf_get_addr(buf_ptr, &buf_addr);

			raw_to_icmp(buf_addr, &hdr, &payload);

			if (hdr->l4_hdr.type == ICMP_ECHO_REQUEST && hdr->l4_hdr.code == 0) {
				icmp_swap_mac_addr(hdr);
				icmp_swap_ip_addr(hdr);
				hdr->l3_hdr.time_to_live = 128;
				hdr->l3_hdr.hdr_checksum = 0;
				nbytes = BYTE_SWAP16(hdr->l3_hdr.total_length) + sizeof(struct ether_hdr);

				hdr->l4_hdr.type = ICMP_ECHO_REPLY;
				hdr->l4_hdr.cksum = 0;
				hdr->l4_hdr.cksum = icmp_checksum((uint16_t *)&(hdr->l4_hdr), nbytes - (sizeof(struct ether_hdr) - sizeof(struct ipv4_hdr)));
				/* Will translate in a notification caught by DOCA PE on the CPU side. */
				doca_gpu_dev_eth_txq_send_enqueue_strong(txq, buf_ptr, nbytes, DOCA_GPU_SEND_FLAG_NOTIFY);
			}
			else
				printf("Unknown ICMP type %d code %d id %d seq %d\n", hdr->l4_hdr.type, hdr->l4_hdr.code, BYTE_SWAP16(hdr->l4_hdr.ident), BYTE_SWAP16(hdr->l4_hdr.seq_nb));

			buf_idx += WARP_SIZE;
		}
		__syncwarp();

		if (lane_id == 0) {
			doca_gpu_dev_eth_txq_commit_strong(txq);
			doca_gpu_dev_eth_txq_push(txq);
		}
		__syncwarp();
	}
}

extern "C" {

doca_error_t kernel_receive_icmp(hipStream_t stream, uint32_t *exit_cond, struct rxq_icmp_queues *icmp_queues)
{
	hipError_t result = hipSuccess;

	if (exit_cond == 0 || icmp_queues == NULL || icmp_queues->numq == 0 || icmp_queues->numq > MAX_QUEUES_ICMP) {
		DOCA_LOG_ERR("kernel_receive_icmp invalid input values");
		return DOCA_ERROR_INVALID_VALUE;
	}

	/* Check no previous CUDA errors */
	result = hipGetLastError();
	if (hipSuccess != result) {
		DOCA_LOG_ERR("[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, hipGetErrorString(result));
		return DOCA_ERROR_BAD_STATE;
	}

	/* Assume MAX_QUEUES_ICMP == 1 */
	cuda_kernel_receive_icmp<<<1, WARP_SIZE, 0, stream>>>(exit_cond, icmp_queues->eth_rxq_gpu[0], icmp_queues->eth_txq_gpu[0]);
	result = hipGetLastError();
	if (hipSuccess != result) {
		DOCA_LOG_ERR("[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, hipGetErrorString(result));
		return DOCA_ERROR_BAD_STATE;
	}

	return DOCA_SUCCESS;
}

} /* extern C */
