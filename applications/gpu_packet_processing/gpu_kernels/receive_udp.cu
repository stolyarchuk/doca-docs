#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023 NVIDIA CORPORATION AND AFFILIATES.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdlib.h>
#include <string.h>

#include <doca_gpunetio_dev_buf.cuh>
#include <doca_gpunetio_dev_sem.cuh>
#include <doca_gpunetio_dev_eth_rxq.cuh>

#include "common.h"
#include "packets.h"
#include "filters.cuh"

#define UDP_WARP_MODE 0

DOCA_LOG_REGISTER(GPU_SANITY::KernelReceiveUdp);

__global__ void cuda_kernel_receive_udp(uint32_t *exit_cond,
					struct doca_gpu_eth_rxq *rxq0, struct doca_gpu_eth_rxq *rxq1, struct doca_gpu_eth_rxq *rxq2, struct doca_gpu_eth_rxq *rxq3,
					int sem_num,
					struct doca_gpu_semaphore_gpu *sem0, struct doca_gpu_semaphore_gpu *sem1, struct doca_gpu_semaphore_gpu *sem2, struct doca_gpu_semaphore_gpu *sem3
				)
{
	__shared__ uint32_t rx_pkt_num;
	__shared__ uint64_t rx_buf_idx;
	__shared__ struct stats_udp stats_sh;

	doca_error_t ret;
	struct doca_gpu_eth_rxq *rxq = NULL;
	struct doca_gpu_semaphore_gpu *sem = NULL;
	struct doca_gpu_buf *buf_ptr;
	struct stats_udp stats_thread;
	struct stats_udp *stats_global;
	struct eth_ip_udp_hdr *hdr;
	uintptr_t buf_addr;
	uint64_t buf_idx = 0;
	uint32_t lane_id = threadIdx.x % WARP_SIZE;
	uint8_t *payload;
	uint32_t sem_idx = 0;

	if (blockIdx.x == 0) {
		rxq = rxq0;
		sem = sem0;
	} else if (blockIdx.x == 1) {
		rxq = rxq1;
		sem = sem1;
	} else if (blockIdx.x == 2) {
		rxq = rxq2;
		sem = sem2;
	} else if (blockIdx.x == 3) {
		rxq = rxq3;
		sem = sem3;
	}
	else
		return;

	if (threadIdx.x == 0) {
		DOCA_GPUNETIO_VOLATILE(stats_sh.dns) = 0;
		DOCA_GPUNETIO_VOLATILE(stats_sh.others) = 0;
	}
	__syncthreads();

	while (DOCA_GPUNETIO_VOLATILE(*exit_cond) == 0) {
		stats_thread.dns = 0;
		stats_thread.others = 0;

		/* No need to impose packet limit here as we want the max number of packets every time */
		ret = doca_gpu_dev_eth_rxq_receive_block(rxq, 0, MAX_RX_TIMEOUT_NS, &rx_pkt_num, &rx_buf_idx);
		/* If any thread returns receive error, the whole execution stops */
		if (ret != DOCA_SUCCESS) {
			if (threadIdx.x == 0) {
				/*
				 * printf in CUDA kernel may be a good idea only to report critical errors or debugging.
				 * If application prints this message on the console, something bad happened and
				 * applications needs to exit
				 */
				printf("Receive UDP kernel error %d Block %d rxpkts %d error %d\n", ret, blockIdx.x, rx_pkt_num, ret);
				DOCA_GPUNETIO_VOLATILE(*exit_cond) = 1;
			}
			break;
		}

		if (rx_pkt_num == 0)
			continue;

		buf_idx = threadIdx.x;
		while (buf_idx < rx_pkt_num) {
			doca_gpu_dev_eth_rxq_get_buf(rxq, rx_buf_idx + buf_idx, &buf_ptr);
			doca_gpu_dev_buf_get_addr(buf_ptr, &buf_addr);
			raw_to_udp(buf_addr, &hdr, &payload);

			if (filter_is_dns(&(hdr->l4_hdr), payload))
				stats_thread.dns++;
			else
				stats_thread.others++;

			/* Double-proof it's not reading old packets */
			wipe_packet_32b((uint8_t*)&(hdr->l4_hdr));
			buf_idx += blockDim.x;
		}
		__syncthreads();

#pragma unroll
		for (int offset = 16; offset > 0; offset /= 2) {
			stats_thread.dns += __shfl_down_sync(WARP_FULL_MASK, stats_thread.dns, offset);
			stats_thread.others += __shfl_down_sync(WARP_FULL_MASK, stats_thread.others, offset);
			__syncwarp();
		}

		if (lane_id == 0) {
			atomicAdd_block((uint32_t *)&(stats_sh.dns), stats_thread.dns);
			atomicAdd_block((uint32_t *)&(stats_sh.others), stats_thread.others);
		}
		__syncthreads();

		if (threadIdx.x == 0 && rx_pkt_num > 0) {
			ret = doca_gpu_dev_semaphore_get_custom_info_addr(sem, sem_idx, (void **)&stats_global);
			if (ret != DOCA_SUCCESS) {
				printf("UDP Error %d doca_gpu_dev_semaphore_get_custom_info_addr block %d thread %d\n", ret, blockIdx.x, threadIdx.x);
				DOCA_GPUNETIO_VOLATILE(*exit_cond) = 1;
				break;
			}

			DOCA_GPUNETIO_VOLATILE(stats_global->dns) = DOCA_GPUNETIO_VOLATILE(stats_sh.dns);
			DOCA_GPUNETIO_VOLATILE(stats_global->others) = DOCA_GPUNETIO_VOLATILE(stats_sh.others);
			DOCA_GPUNETIO_VOLATILE(stats_global->total) = rx_pkt_num;
			doca_gpu_dev_semaphore_set_status(sem, sem_idx, DOCA_GPU_SEMAPHORE_STATUS_READY);
			__threadfence_system();

			sem_idx = (sem_idx + 1) % sem_num;

			DOCA_GPUNETIO_VOLATILE(stats_sh.dns) = 0;
			DOCA_GPUNETIO_VOLATILE(stats_sh.others) = 0;
		}

		__syncthreads();
	}
}

extern "C" {

doca_error_t kernel_receive_udp(hipStream_t stream, uint32_t *exit_cond, struct rxq_udp_queues *udp_queues)
{
	hipError_t result = hipSuccess;

	if (udp_queues == NULL || udp_queues->numq == 0 || udp_queues->numq > MAX_QUEUES || exit_cond == 0) {
		DOCA_LOG_ERR("kernel_receive_udp invalid input values");
		return DOCA_ERROR_INVALID_VALUE;
	}

	/* Check no previous CUDA errors */
	result = hipGetLastError();
	if (hipSuccess != result) {
		DOCA_LOG_ERR("[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, hipGetErrorString(result));
		return DOCA_ERROR_BAD_STATE;
	}

	/* Assume MAX_QUEUES == 4 */
	cuda_kernel_receive_udp<<<udp_queues->numq, CUDA_THREADS, 0, stream>>>(exit_cond,
									udp_queues->eth_rxq_gpu[0], udp_queues->eth_rxq_gpu[1], udp_queues->eth_rxq_gpu[2], udp_queues->eth_rxq_gpu[3],
									udp_queues->nums,
									udp_queues->sem_gpu[0], udp_queues->sem_gpu[1], udp_queues->sem_gpu[2], udp_queues->sem_gpu[3]
									);
	result = hipGetLastError();
	if (hipSuccess != result) {
		DOCA_LOG_ERR("[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, hipGetErrorString(result));
		return DOCA_ERROR_BAD_STATE;
	}

	return DOCA_SUCCESS;
}

} /* extern C */
